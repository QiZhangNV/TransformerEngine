#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hipblaslt.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <transformer_engine/gemm.h>
#include <transformer_engine/transformer_engine.h>

#include <cstdint>
#include <mutex>
#include <vector>

#include "../common.h"
#include "../util/handle_manager.h"
#include "../util/logging.h"
#include "common/util/hip/hip_runtime.h"
#include "cute/tensor.hpp"
#include "cutlass/cutlass.h"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/group_array_problem_shape.hpp"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/util/device_memory.h"

using namespace cute;

/*****
 * Fprop and Dgrad
 ******/

template <typename Sm1xxBlkScaledConfig, typename UnderlyingProblemShape, typename ElementA,
          typename ElementD, typename ElementSF, typename StrideA, typename StrideB,
          typename StrideD, typename LayoutSFA, typename LayoutSFB, bool transB>
__global__ void setGroupedGemmArguments(int num_experts, const int64_t *gemm_m_per_expert,
                                        int gemm_n, int gemm_k, ElementA *ptr_A, ElementSF *ptr_SFA,
                                        ElementD *ptr_D, UnderlyingProblemShape *problem_sizes,
                                        ElementA **ptr_A_list, ElementSF **ptr_SFA_list,
                                        StrideA *stride_A_list, LayoutSFA *layout_SFA_list,
                                        StrideB *stride_B_list, LayoutSFB *layout_SFB_list,
                                        ElementD **ptr_D_list, StrideD *stride_D_list) {
  int m_offset = 0;
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    for (int expert_id = 0; expert_id < num_experts; expert_id++) {
      int gemm_m = int(gemm_m_per_expert[expert_id]);
      problem_sizes[expert_id] = cute::make_shape(gemm_m, gemm_n, gemm_k);
      // printf("problem_sizes: %d, %d, %d\n", gemm_m, gemm_n, gemm_k);

      ptr_A_list[expert_id] = ptr_A + m_offset * gemm_k;
      ptr_SFA_list[expert_id] = ptr_SFA + m_offset * ((gemm_k + 127) / 128 * 4);
      stride_A_list[expert_id] = cute::make_stride(int64_t(gemm_k), _1{}, _0{});
      layout_SFA_list[expert_id] =
          Sm1xxBlkScaledConfig::tile_atom_to_shape_SFA(cute::make_shape(gemm_m, gemm_n, gemm_k, 1));

      if constexpr (transB) {
        stride_B_list[expert_id] = cute::make_stride(int64_t(gemm_k), _1{}, _0{});
      } else {
        stride_B_list[expert_id] = cute::make_stride(_1{}, int64_t(gemm_n), _0{});
      }
      layout_SFB_list[expert_id] =
          Sm1xxBlkScaledConfig::tile_atom_to_shape_SFB(cute::make_shape(gemm_m, gemm_n, gemm_k, 1));

      ptr_D_list[expert_id] = ptr_D + m_offset * gemm_n;
      stride_D_list[expert_id] = cute::make_stride(int64_t(gemm_n), _1{}, _0{});

      m_offset += gemm_m;
    }
  }
}

template <typename T, typename TSF, typename WeightType, typename WeightTypeSF, typename OutputType,
          bool TransB>
void generic_moe_gemm_kernelLauncher(T *A, TSF *SFA, WeightType **B_list, WeightTypeSF **SFB_list,
                                     OutputType *D, const int64_t *gemm_m_per_expert, int gemm_n,
                                     int gemm_k, int num_experts, size_t workspaceSize,
                                     void *workspace, hipStream_t stream,
                                     int *kernel_occupancy = nullptr) {
  using ProblemShape = cutlass::gemm::GroupProblemShape<Shape<int, int, int>>;  // <M,N,K> per group
  using ElementInput = cutlass::float_e4m3_t;  // Element type for Input matrix operands
  using ElementSF = cutlass::float_ue8m0_t;    // Element type for SF matrix operands
  using ElementC = cutlass::bfloat16_t;

  using ElementA = cutlass::mx_float8_t<ElementInput>;  // Element type for A matrix operand
  using LayoutA = cutlass::layout::RowMajor;            // Layout type for A matrix operand
  constexpr int AlignmentA = 32;  // Alignment of A matrix in units of elements (up to 16 bytes)

  // B matrix configuration
  using ElementB = cutlass::mx_float8_t<ElementInput>;  // Element type for B matrix operand
  using LayoutB =
      cute::conditional_t<TransB, cutlass::layout::ColumnMajor,
                          cutlass::layout::RowMajor>;  // Layout type for B matrix operand
  constexpr int AlignmentB = 32;  // Alignment of A matrix in units of elements (up to 16 bytes)

  // C/D matrix configuration
  using ElementD = ElementC;                  // Element type for D matrix operands
  using LayoutC = cutlass::layout::RowMajor;  // Layout type for C and D matrix operands
  constexpr int AlignmentC =
      128 / cutlass::sizeof_bits<
                ElementC>::value;  // Alignment of C matrix in units of elements (up to 16 bytes)
  constexpr int AlignmentD =
      128 / cutlass::sizeof_bits<
                ElementD>::value;    // Alignment of D matrix in units of elements (up to 16 bytes)
  using ElementAccumulator = float;  // Element type for internal accumulation

  // Core kernel configurations
  using ArchTag =
      cutlass::arch::Sm100;  // Tag indicating the minimum SM that supports the intended feature
  using EpilogueOperatorClass = cutlass::arch::OpClassTensorOp;  // Epilogue Operator class tag
  using MainloopOperatorClass =
      cutlass::arch::OpClassBlockScaledTensorOp;  // Mainloop Operator class tag
  using StageCountType =
      cutlass::gemm::collective::StageCountAuto;  // Stage count maximized based on the tile size

  // Runtime Cluster Shape
  using ClusterShape = Shape<int32_t, int32_t, _1>;

  struct MMA2SMConfig {
    using MmaTileShape = Shape<_256, _256, _128>;
    using KernelSchedule =
        cutlass::gemm::KernelPtrArrayTmaWarpSpecialized2SmMxf8f6f4Sm100;  // Kernel to launch
    using EpilogueSchedule =
        cutlass::epilogue::PtrArrayTmaWarpSpecialized2Sm;  // Epilogue to launch
  };

  using CollectiveEpilogue2SM = typename cutlass::epilogue::collective::CollectiveBuilder<
      ArchTag, EpilogueOperatorClass, typename MMA2SMConfig::MmaTileShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto, ElementAccumulator, ElementAccumulator, void,
      LayoutC *, AlignmentC, ElementD, LayoutC *, AlignmentD,
      typename MMA2SMConfig::EpilogueSchedule
      // , FusionOperation  // Enable for SF Output
      >::CollectiveOp;
  using CollectiveMainloop2SM = typename cutlass::gemm::collective::CollectiveBuilder<
      ArchTag, MainloopOperatorClass, ElementA, LayoutA *, AlignmentA, ElementB, LayoutB *,
      AlignmentB, ElementAccumulator, typename MMA2SMConfig::MmaTileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
          sizeof(typename CollectiveEpilogue2SM::SharedStorage))>,
      typename MMA2SMConfig::KernelSchedule>::CollectiveOp;
  using GemmKernel2SM = cutlass::gemm::kernel::GemmUniversal<ProblemShape, CollectiveMainloop2SM,
                                                             CollectiveEpilogue2SM>;
  using GemmGrouped = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel2SM>;

  using StrideA = typename GemmGrouped::GemmKernel::InternalStrideA;
  using StrideB = typename GemmGrouped::GemmKernel::InternalStrideB;
  using StrideC = typename GemmGrouped::GemmKernel::InternalStrideC;
  using StrideD = typename GemmGrouped::GemmKernel::InternalStrideD;

  using LayoutSFA = typename GemmGrouped::GemmKernel::CollectiveMainloop::InternalLayoutSFA;
  using LayoutSFB = typename GemmGrouped::GemmKernel::CollectiveMainloop::InternalLayoutSFB;
  using Sm1xxBlkScaledConfig =
      typename GemmGrouped::GemmKernel::CollectiveMainloop::Sm1xxBlkScaledConfig;

  using RasterOrderOptions = cutlass::gemm::kernel::detail::RasterOrderOptions;

  auto get_aligned_offset = [](size_t current_offset, size_t alignment) -> size_t {
    return (current_offset + alignment - 1) & ~(alignment - 1);
  };

  if (workspace == nullptr) {
    throw std::runtime_error("[FT Error][MoE Runner] workspace is null");
  }

  size_t offset = 0;
  typename GemmGrouped::ElementA *ptr_A = reinterpret_cast<typename GemmGrouped::ElementA *>(A);
  typename GemmGrouped::ElementA **ptr_A_list = reinterpret_cast<typename GemmGrouped::ElementA **>(
      reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(offset + num_experts * sizeof(typename GemmGrouped::ElementA *), 128);

  typename GemmGrouped::ElementB **ptr_B_list = reinterpret_cast<typename GemmGrouped::ElementB **>(
      reinterpret_cast<char *>(workspace) + offset);
  hipMemcpyAsync(ptr_B_list, B_list, num_experts * sizeof(typename GemmGrouped::ElementB *),
                  hipMemcpyHostToDevice, stream);
  offset = get_aligned_offset(offset + num_experts * sizeof(typename GemmGrouped::ElementB *), 128);

  typename GemmGrouped::ElementD *ptr_D = reinterpret_cast<typename GemmGrouped::ElementD *>(D);
  typename GemmGrouped::ElementD **ptr_D_list = reinterpret_cast<typename GemmGrouped::ElementD **>(
      reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(offset + num_experts * sizeof(typename GemmGrouped::ElementD *), 128);

  typename GemmGrouped::GemmKernel::ElementSF *ptr_SFA =
      reinterpret_cast<typename GemmGrouped::GemmKernel::ElementSF *>(SFA);
  typename GemmGrouped::GemmKernel::ElementSF **ptr_SFA_list =
      reinterpret_cast<typename GemmGrouped::GemmKernel::ElementSF **>(
          reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(
      offset + num_experts * sizeof(typename GemmGrouped::GemmKernel::ElementSF *), 128);

  typename GemmGrouped::GemmKernel::ElementSF **ptr_SFB_list =
      reinterpret_cast<typename GemmGrouped::GemmKernel::ElementSF **>(
          reinterpret_cast<char *>(workspace) + offset);
  hipMemcpyAsync(ptr_SFB_list, SFB_list,
                  num_experts * sizeof(typename GemmGrouped::GemmKernel::ElementSF *),
                  hipMemcpyHostToDevice, stream);
  offset = get_aligned_offset(
      offset + num_experts * sizeof(typename GemmGrouped::GemmKernel::ElementSF *), 128);

  StrideA *stride_A_list =
      reinterpret_cast<StrideA *>(reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(offset + num_experts * sizeof(StrideA), 128);

  StrideB *stride_B_list =
      reinterpret_cast<StrideB *>(reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(offset + num_experts * sizeof(StrideB), 128);

  StrideD *stride_D_list =
      reinterpret_cast<StrideD *>(reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(offset + num_experts * sizeof(StrideD), 128);

  LayoutSFA *layout_SFA_list =
      reinterpret_cast<LayoutSFA *>(reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(offset + num_experts * sizeof(LayoutSFA), 128);

  LayoutSFB *layout_SFB_list =
      reinterpret_cast<LayoutSFB *>(reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(offset + num_experts * sizeof(LayoutSFB), 128);

  ProblemShape::UnderlyingProblemShape *problem_sizes =
      reinterpret_cast<ProblemShape::UnderlyingProblemShape *>(reinterpret_cast<char *>(workspace) +
                                                               offset);
  offset =
      get_aligned_offset(offset + num_experts * sizeof(ProblemShape::UnderlyingProblemShape), 128);

  setGroupedGemmArguments<Sm1xxBlkScaledConfig, ProblemShape::UnderlyingProblemShape,
                          typename GemmGrouped::ElementA, typename GemmGrouped::ElementD,
                          typename GemmGrouped::GemmKernel::ElementSF, StrideA, StrideB, StrideD,
                          LayoutSFA, LayoutSFB, TransB><<<1, 32, 0, stream>>>(
      num_experts, gemm_m_per_expert, gemm_n, gemm_k, ptr_A, ptr_SFA, ptr_D, problem_sizes,
      ptr_A_list, ptr_SFA_list, stride_A_list, layout_SFA_list, stride_B_list, layout_SFB_list,
      ptr_D_list, stride_D_list);

  typename GemmGrouped::Arguments args;
  decltype(args.epilogue.thread) fusion_args;
  fusion_args.alpha_ptr = nullptr;
  fusion_args.beta_ptr = nullptr;
  // Set alpha and beta to 1 and 0 for the fusion operation
  fusion_args.alpha = 1;
  fusion_args.alpha_ptr_array = nullptr;
  fusion_args.dAlpha = {_0{}, _0{}, 0};
  fusion_args.beta = 0;
  fusion_args.beta_ptr_array = nullptr;
  fusion_args.dBeta = {_0{}, _0{}, 0};

  cutlass::KernelHardwareInfo hw_info;
  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  hw_info.sm_count =
      cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  if (!is_static_v<ClusterShape>) {
    hw_info.cluster_shape = dim3(4, 4, 1);
    hw_info.cluster_shape_fallback = dim3(2, 1, 1);
  }

  typename GemmGrouped::GemmKernel::TileSchedulerArguments scheduler;
  scheduler.raster_order = RasterOrderOptions::AlongN;

  args = typename GemmGrouped::Arguments{
      cutlass::gemm::GemmUniversalMode::kGrouped,
      {num_experts, problem_sizes, nullptr},
      {const_cast<const typename GemmGrouped::ElementA **>(ptr_A_list), stride_A_list,
       const_cast<const typename GemmGrouped::ElementB **>(ptr_B_list), stride_B_list,
       const_cast<const typename GemmGrouped::GemmKernel::ElementSF **>(ptr_SFA_list),
       layout_SFA_list,
       const_cast<const typename GemmGrouped::GemmKernel::ElementSF **>(ptr_SFB_list),
       layout_SFB_list},
      {fusion_args, nullptr, stride_D_list, ptr_D_list, stride_D_list},
      hw_info,
      scheduler};

  GemmGrouped gemm;

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = GemmGrouped::get_workspace_size(args);
  if (workspaceSize < offset + workspace_size) {  // 16MB limit
    throw std::runtime_error("Calculated workspace size (" +
                             std::to_string(offset + workspace_size) + ") exceeds buffer size (" +
                             std::to_string(workspaceSize) + ")\n");
  }

  auto can_implement = gemm.can_implement(args);
  if (can_implement != cutlass::Status::kSuccess) {
    std::string err_msg = "MoE kernel will fail for params. Error: " +
                          std::string(cutlassGetStatusString(can_implement));
    throw std::runtime_error("[FT Error][MoE Runner] " + err_msg);
  }

  auto init_status = gemm.initialize(args, reinterpret_cast<char *>(workspace) + offset);
  if (init_status != cutlass::Status::kSuccess) {
    std::string err_msg = "Failed to initialize cutlass grouped gemm. Error: " +
                          std::string(cutlassGetStatusString(init_status));
    throw std::runtime_error("[FT Error][MoE Runner] " + err_msg);
  }

  auto run_status = gemm.run(stream);
  if (run_status != cutlass::Status::kSuccess) {
    std::string err_msg = "Failed to run cutlass grouped gemm. Error: " +
                          std::string(cutlassGetStatusString(run_status));
    throw std::runtime_error("[FT Error][MoE Runner] " + err_msg);
  }
}

// Only mxfp8 is supported for now
// A is single Tensor, B is splited tensor list, D is single Tensor
void nvte_cutlass_grouped_gemm(const NVTETensor *A, const NVTETensor *B, NVTETensor *D,
                               const int64_t *m_splits, const NVTETensor *bias,
                               NVTETensor *pre_gelu_out, const int num_gemms, bool transa,
                               bool transb, bool grad, NVTETensor *workspace, size_t workspaceSize,
                               bool accumulate, bool use_split_accumulator, int math_sm_count,
                               hipStream_t stream) {
  NVTE_API_CALL(nvte_cutlass_grouped_gemm);
  using namespace transformer_engine;
  // printf("===========nvte_cutlass_grouped_gemm===========\n");
  // printf("transa: %d, transb: %d\n", transa, transb);
  // printf("grad: %d\n", grad);

  // Process A
  const transformer_engine::Tensor *inputA = convertNVTETensor(A[0]);
  if (transa) {
    NVTE_CHECK(inputA->has_columnwise_data(), "Input A is missing column-wise usage");
  } else {
    NVTE_CHECK(inputA->has_data(), "Input A is missing row-wise usage");
  }
  __hip_fp8_e4m3_fnuz *inputA_ptr = transa
                                  ? reinterpret_cast<__hip_fp8_e4m3_fnuz *>(inputA->columnwise_data.dptr)
                                  : reinterpret_cast<__hip_fp8_e4m3_fnuz *>(inputA->data.dptr);
  __nv_fp8_e8m0 *inputA_SF_ptr =
      transa ? reinterpret_cast<__nv_fp8_e8m0 *>(inputA->columnwise_scale_inv.dptr)
             : reinterpret_cast<__nv_fp8_e8m0 *>(inputA->scale_inv.dptr);

  // Process B
  __hip_fp8_e4m3_fnuz *inputB_ptr_list[num_gemms];
  __nv_fp8_e8m0 *inputB_SF_ptr_list[num_gemms];
  for (size_t i = 0; i < num_gemms; i++) {
    const transformer_engine::Tensor *inputB = convertNVTETensor(B[i]);
    if (transb) {
      NVTE_CHECK(inputB->has_data(), "Input B is missing row-wise usage");
    } else {
      NVTE_CHECK(inputB->has_columnwise_data(), "Input B is missing column-wise usage");
    }
    inputB_ptr_list[i] = transb ? reinterpret_cast<__hip_fp8_e4m3_fnuz *>(inputB->data.dptr)
                                : reinterpret_cast<__hip_fp8_e4m3_fnuz *>(inputB->columnwise_data.dptr);
    inputB_SF_ptr_list[i] =
        transb ? reinterpret_cast<__nv_fp8_e8m0 *>(inputB->scale_inv.dptr)
               : reinterpret_cast<__nv_fp8_e8m0 *>(inputB->columnwise_scale_inv.dptr);
  }

  // Process D
  const transformer_engine::Tensor *outputD = convertNVTETensor(D[0]);
  NVTE_CHECK(outputD->has_data(), "Input D is missing row-wise usage");
  __hip_bfloat16 *outputD_ptr = reinterpret_cast<__hip_bfloat16 *>(outputD->data.dptr);

  // Get GEMM shape
  const int gemm_k = transa ? inputA->flat_first_dim() : inputA->flat_last_dim();
  const int gemm_n =
      transb ? convertNVTETensor(B[0])->flat_first_dim() : convertNVTETensor(B[0])->flat_last_dim();
  //   printf("num_gemms: %d\n", num_gemms);
  //   printf("gemm_n: %d, gemm_k: %d\n", gemm_n, gemm_k);
  if ((gemm_k & 0x1F) != 0) {
    throw std::runtime_error("gemm_k of grouped gemm with variable M must be a multiple of 32.");
  }

  if (transb) {
    generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __nv_fp8_e8m0, __hip_fp8_e4m3_fnuz, __nv_fp8_e8m0,
                                    __hip_bfloat16, true>(
        inputA_ptr, inputA_SF_ptr, inputB_ptr_list, inputB_SF_ptr_list, outputD_ptr,
        m_splits,  // gemm_m splits
        gemm_n,    // gemm_n
        gemm_k,    // gemm_k
        num_gemms, workspaceSize, convertNVTETensor(workspace[0])->data.dptr, stream);
  } else {
    generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __nv_fp8_e8m0, __hip_fp8_e4m3_fnuz, __nv_fp8_e8m0,
                                    __hip_bfloat16, false>(
        inputA_ptr, inputA_SF_ptr, inputB_ptr_list, inputB_SF_ptr_list, outputD_ptr,
        m_splits,  // gemm_m splits
        gemm_n,    // gemm_n
        gemm_k,    // gemm_k
        num_gemms, workspaceSize, convertNVTETensor(workspace[0])->data.dptr, stream);
  }
}

/*****
 * Wgrad
 ******/

template <typename Sm1xxBlkScaledConfig, typename UnderlyingProblemShape, typename ElementA,
          typename ElementB, typename ElementD, typename ElementSF, typename StrideA,
          typename StrideB, typename StrideD, typename LayoutSFA, typename LayoutSFB, bool transD>
__global__ void setGroupedGemmWgradArguments(
    int num_experts, int gemm_m, int gemm_n, const int64_t *gemm_k_per_expert, int total_gemm_k,
    ElementA *ptr_A, ElementSF *ptr_SFA, ElementB *ptr_B, ElementSF *ptr_SFB,
    UnderlyingProblemShape *problem_sizes, ElementA **ptr_A_list, ElementSF **ptr_SFA_list,
    StrideA *stride_A_list, LayoutSFA *layout_SFA_list, ElementB **ptr_B_list,
    ElementSF **ptr_SFB_list, StrideB *stride_B_list, LayoutSFB *layout_SFB_list,
    ElementD **ptr_D_list, StrideD *stride_D_list, bool accumulate_D) {
  // printf("===========wgrad setGroupedGemmWgradArguments===========\n");
  // printf("transD: %d\n", transD);
  int k_offset = 0;
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    for (int expert_id = 0; expert_id < num_experts; expert_id++) {
      int gemm_k = int(gemm_k_per_expert[expert_id]);
      if (gemm_k == 0) {
        // If gemm_k is 0, we need to set the problem_sizes to 0, 0, 0 to skip the gemm
        problem_sizes[expert_id] = cute::make_shape(0, 0, 0);
        if (!accumulate_D) {
          for (int i = 0; i < gemm_m * gemm_n; i++) {
            ptr_D_list[expert_id][i] = ElementD(0);
          }
        }
        continue;
      }
      problem_sizes[expert_id] = cute::make_shape(gemm_m, gemm_n, gemm_k);
      // printf("wgrad problem_sizes: %d, %d, %d\n", gemm_m, gemm_n, gemm_k);

      ptr_A_list[expert_id] = ptr_A + gemm_m * k_offset;
      ptr_SFA_list[expert_id] = ptr_SFA + 128 * ((k_offset + 127) / 128 * 4);
      stride_A_list[expert_id] = cute::make_stride(_1{}, int64_t(gemm_m), _0{});
      auto temp_sfa_layout = Sm1xxBlkScaledConfig::tile_atom_to_shape_SFA(
          cute::make_shape(gemm_m, gemm_n, total_gemm_k, 1));
      layout_SFA_list[expert_id] = cute::make_layout(
          get<0>(temp_sfa_layout),
          make_layout(get<0>(get<1>(temp_sfa_layout)),
                      make_layout(gemm_k / 128, get<1>(get<1>(temp_sfa_layout.stride())))),

          get<2>(temp_sfa_layout));

      ptr_B_list[expert_id] = ptr_B + gemm_n * k_offset;
      ptr_SFB_list[expert_id] = ptr_SFB + 128 * ((k_offset + 127) / 128 * 4);
      stride_B_list[expert_id] = cute::make_stride(_1{}, int64_t(gemm_n), _0{});
      auto temp_sfb_layout = Sm1xxBlkScaledConfig::tile_atom_to_shape_SFB(
          cute::make_shape(gemm_m, gemm_n, total_gemm_k, 1));
      layout_SFB_list[expert_id] = cute::make_layout(
          get<0>(temp_sfb_layout),
          make_layout(get<0>(get<1>(temp_sfb_layout)),
                      make_layout(gemm_k / 128, get<1>(get<1>(temp_sfb_layout.stride())))),
          get<2>(temp_sfb_layout));

      if constexpr (transD) {
        stride_D_list[expert_id] = cute::make_stride(_1{}, int64_t(gemm_m), _0{});
      } else {
        stride_D_list[expert_id] = cute::make_stride(int64_t(gemm_n), _1{}, _0{});
      }

      k_offset += gemm_k;
    }
  }
}

template <typename T, typename TSF, typename WeightType, typename WeightTypeSF, typename OutputType,
          bool TransD>
void generic_moe_gemm_wgrad_kernelLauncher(T *A, TSF *SFA, WeightType *B, WeightTypeSF *SFB,
                                           void **D_list, int gemm_m, int gemm_n,
                                           const int64_t *gemm_k_per_expert, int total_gemm_k,
                                           int num_experts, bool accumulate_D, size_t workspaceSize,
                                           void *workspace, hipStream_t stream,
                                           int *kernel_occupancy = nullptr) {
  using ProblemShape = cutlass::gemm::GroupProblemShape<Shape<int, int, int>>;  // <M,N,K> per group
  using ElementInput = cutlass::float_e4m3_t;  // Element type for Input matrix operands
  using ElementSF = cutlass::float_ue8m0_t;    // Element type for SF matrix operands
  using ElementC =
      cute::conditional_t<cute::is_same_v<OutputType, __hip_bfloat16>, cutlass::bfloat16_t, float>;

  using ElementA = cutlass::mx_float8_t<ElementInput>;  // Element type for A matrix operand
  using LayoutA = cutlass::layout::ColumnMajor;         // Layout type for A matrix operand
  constexpr int AlignmentA = 32;  // Alignment of A matrix in units of elements (up to 16 bytes)

  // B matrix configuration
  using ElementB = cutlass::mx_float8_t<ElementInput>;  // Element type for B matrix operand
  using LayoutB = cutlass::layout::RowMajor;            // Layout type for B matrix operand
  constexpr int AlignmentB = 32;  // Alignment of A matrix in units of elements (up to 16 bytes)

  // C/D matrix configuration
  using ElementD = ElementC;  // Element type for D matrix operands
  using LayoutC = typename cutlass::platform::conditional<
      TransD, cutlass::layout::ColumnMajor,
      cutlass::layout::RowMajor>::type;  // Layout type for C and D matrix operands
  constexpr int AlignmentC =
      128 / cutlass::sizeof_bits<
                ElementC>::value;  // Alignment of C matrix in units of elements (up to 16 bytes)
  constexpr int AlignmentD =
      128 / cutlass::sizeof_bits<
                ElementD>::value;    // Alignment of D matrix in units of elements (up to 16 bytes)
  using ElementAccumulator = float;  // Element type for internal accumulation

  // Core kernel configurations
  using ArchTag =
      cutlass::arch::Sm100;  // Tag indicating the minimum SM that supports the intended feature
  using EpilogueOperatorClass = cutlass::arch::OpClassTensorOp;  // Epilogue Operator class tag
  using MainloopOperatorClass =
      cutlass::arch::OpClassBlockScaledTensorOp;  // Mainloop Operator class tag
  using StageCountType =
      cutlass::gemm::collective::StageCountAuto;  // Stage count maximized based on the tile size

  // Runtime Cluster Shape
  using ClusterShape = Shape<int32_t, int32_t, _1>;

  struct MMA2SMConfig {
    using MmaTileShape = Shape<_256, _256, _128>;
    using KernelSchedule =
        cutlass::gemm::KernelPtrArrayTmaWarpSpecialized2SmMxf8f6f4Sm100;  // Kernel to launch
    using EpilogueSchedule =
        cutlass::epilogue::PtrArrayTmaWarpSpecialized2Sm;  // Epilogue to launch
  };

  using CollectiveEpilogue2SM = typename cutlass::epilogue::collective::CollectiveBuilder<
      ArchTag, EpilogueOperatorClass, typename MMA2SMConfig::MmaTileShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto, ElementAccumulator, ElementAccumulator,
      ElementC, LayoutC *, AlignmentC, ElementD, LayoutC *, AlignmentD,
      typename MMA2SMConfig::EpilogueSchedule
      // , FusionOperation  // Enable for SF Output
      >::CollectiveOp;
  using CollectiveMainloop2SM = typename cutlass::gemm::collective::CollectiveBuilder<
      ArchTag, MainloopOperatorClass, ElementA, LayoutA *, AlignmentA, ElementB, LayoutB *,
      AlignmentB, ElementAccumulator, typename MMA2SMConfig::MmaTileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
          sizeof(typename CollectiveEpilogue2SM::SharedStorage))>,
      typename MMA2SMConfig::KernelSchedule>::CollectiveOp;
  using GemmKernel2SM = cutlass::gemm::kernel::GemmUniversal<ProblemShape, CollectiveMainloop2SM,
                                                             CollectiveEpilogue2SM>;
  using GemmGrouped = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel2SM>;

  using StrideA = typename GemmGrouped::GemmKernel::InternalStrideA;
  using StrideB = typename GemmGrouped::GemmKernel::InternalStrideB;
  using StrideC = typename GemmGrouped::GemmKernel::InternalStrideC;
  using StrideD = typename GemmGrouped::GemmKernel::InternalStrideD;

  using LayoutSFA = typename GemmGrouped::GemmKernel::CollectiveMainloop::InternalLayoutSFA;
  using LayoutSFB = typename GemmGrouped::GemmKernel::CollectiveMainloop::InternalLayoutSFB;
  using Sm1xxBlkScaledConfig =
      typename GemmGrouped::GemmKernel::CollectiveMainloop::Sm1xxBlkScaledConfig;

  using RasterOrderOptions = cutlass::gemm::kernel::detail::RasterOrderOptions;

  // Helper function to calculate aligned offset
  auto get_aligned_offset = [](int current_offset, int alignment) -> int {
    return (current_offset + alignment - 1) & ~(alignment - 1);
  };

  if (workspace == nullptr) {
    throw std::runtime_error("[FT Error][MoE Runner] workspace is null");
  }

  int offset = 0;
  typename GemmGrouped::ElementA *ptr_A = reinterpret_cast<typename GemmGrouped::ElementA *>(A);
  typename GemmGrouped::ElementA **ptr_A_list = reinterpret_cast<typename GemmGrouped::ElementA **>(
      reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(offset + num_experts * sizeof(typename GemmGrouped::ElementA *), 128);

  typename GemmGrouped::ElementB *ptr_B = reinterpret_cast<typename GemmGrouped::ElementB *>(B);
  typename GemmGrouped::ElementB **ptr_B_list = reinterpret_cast<typename GemmGrouped::ElementB **>(
      reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(offset + num_experts * sizeof(typename GemmGrouped::ElementB *), 128);

  typename GemmGrouped::ElementD **ptr_D_list = reinterpret_cast<typename GemmGrouped::ElementD **>(
      reinterpret_cast<char *>(workspace) + offset);
  hipMemcpyAsync(ptr_D_list, D_list, num_experts * sizeof(typename GemmGrouped::ElementD *),
                  hipMemcpyHostToDevice, stream);
  offset = get_aligned_offset(offset + num_experts * sizeof(typename GemmGrouped::ElementD *), 128);

  typename GemmGrouped::GemmKernel::ElementSF *ptr_SFA =
      reinterpret_cast<typename GemmGrouped::GemmKernel::ElementSF *>(SFA);
  typename GemmGrouped::GemmKernel::ElementSF **ptr_SFA_list =
      reinterpret_cast<typename GemmGrouped::GemmKernel::ElementSF **>(
          reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(
      offset + num_experts * sizeof(typename GemmGrouped::GemmKernel::ElementSF *), 128);

  typename GemmGrouped::GemmKernel::ElementSF *ptr_SFB =
      reinterpret_cast<typename GemmGrouped::GemmKernel::ElementSF *>(SFB);
  typename GemmGrouped::GemmKernel::ElementSF **ptr_SFB_list =
      reinterpret_cast<typename GemmGrouped::GemmKernel::ElementSF **>(
          reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(
      offset + num_experts * sizeof(typename GemmGrouped::GemmKernel::ElementSF *), 128);

  StrideA *stride_A_list =
      reinterpret_cast<StrideA *>(reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(offset + num_experts * sizeof(StrideA), 128);
  StrideB *stride_B_list =
      reinterpret_cast<StrideB *>(reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(offset + num_experts * sizeof(StrideB), 128);
  StrideD *stride_D_list =
      reinterpret_cast<StrideD *>(reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(offset + num_experts * sizeof(StrideD), 128);

  LayoutSFA *layout_SFA_list =
      reinterpret_cast<LayoutSFA *>(reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(offset + num_experts * sizeof(LayoutSFA), 128);
  LayoutSFB *layout_SFB_list =
      reinterpret_cast<LayoutSFB *>(reinterpret_cast<char *>(workspace) + offset);
  offset = get_aligned_offset(offset + num_experts * sizeof(LayoutSFB), 128);

  ProblemShape::UnderlyingProblemShape *problem_sizes =
      reinterpret_cast<ProblemShape::UnderlyingProblemShape *>(reinterpret_cast<char *>(workspace) +
                                                               offset);
  offset =
      get_aligned_offset(offset + num_experts * sizeof(ProblemShape::UnderlyingProblemShape), 128);

  setGroupedGemmWgradArguments<Sm1xxBlkScaledConfig, ProblemShape::UnderlyingProblemShape,
                               typename GemmGrouped::ElementA, typename GemmGrouped::ElementB,
                               typename GemmGrouped::ElementD,
                               typename GemmGrouped::GemmKernel::ElementSF, StrideA, StrideB,
                               StrideD, LayoutSFA, LayoutSFB, TransD><<<1, 32, 0, stream>>>(
      num_experts, gemm_m, gemm_n, gemm_k_per_expert, total_gemm_k, ptr_A, ptr_SFA, ptr_B, ptr_SFB,
      problem_sizes, ptr_A_list, ptr_SFA_list, stride_A_list, layout_SFA_list, ptr_B_list,
      ptr_SFB_list, stride_B_list, layout_SFB_list, ptr_D_list, stride_D_list, accumulate_D);

  // Check for CUDA errors after kernel launch
  hipError_t cuda_error = hipGetLastError();
  if (cuda_error != hipSuccess) {
    std::string err_msg = "Failed to run setGroupedGemmWgradArguments. CUDA Error: " +
                          std::string(hipGetErrorString(cuda_error));
    throw std::runtime_error("[FT Error][MoE Runner] " + err_msg);
  }

  typename GemmGrouped::Arguments args;
  decltype(args.epilogue.thread) fusion_args;
  fusion_args.alpha_ptr = nullptr;
  fusion_args.beta_ptr = nullptr;
  // Set alpha and beta
  fusion_args.alpha = 1;
  fusion_args.alpha_ptr_array = nullptr;
  fusion_args.dAlpha = {_0{}, _0{}, 0};
  fusion_args.beta = accumulate_D ? 1 : 0;
  fusion_args.beta_ptr_array = nullptr;
  fusion_args.dBeta = {_0{}, _0{}, 0};

  cutlass::KernelHardwareInfo hw_info;
  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  hw_info.sm_count =
      cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  if (!is_static_v<ClusterShape>) {
    hw_info.cluster_shape = dim3(4, 4, 1);
    hw_info.cluster_shape_fallback = dim3(2, 1, 1);
  }

  typename GemmGrouped::GemmKernel::TileSchedulerArguments scheduler;
  scheduler.raster_order = RasterOrderOptions::AlongN;

  args = typename GemmGrouped::Arguments{
      cutlass::gemm::GemmUniversalMode::kGrouped,
      {num_experts, problem_sizes, nullptr},
      {const_cast<const typename GemmGrouped::ElementA **>(ptr_A_list), stride_A_list,
       const_cast<const typename GemmGrouped::ElementB **>(ptr_B_list), stride_B_list,
       const_cast<const typename GemmGrouped::GemmKernel::ElementSF **>(ptr_SFA_list),
       layout_SFA_list,
       const_cast<const typename GemmGrouped::GemmKernel::ElementSF **>(ptr_SFB_list),
       layout_SFB_list},
      {fusion_args,
       accumulate_D ? const_cast<const typename GemmGrouped::ElementC **>(ptr_D_list) : nullptr,
       stride_D_list, ptr_D_list, stride_D_list},
      hw_info,
      scheduler};

  GemmGrouped gemm;

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = GemmGrouped::get_workspace_size(args);
  if (workspaceSize < offset + workspace_size) {  // 16MB limit
    throw std::runtime_error("Calculated workspace size (" +
                             std::to_string(offset + workspace_size) + ") exceeds buffer size (" +
                             std::to_string(workspaceSize) + ")\n");
  }

  auto can_implement = gemm.can_implement(args);
  if (can_implement != cutlass::Status::kSuccess) {
    std::string err_msg = "MoE kernel will fail for params. Error: " +
                          std::string(cutlassGetStatusString(can_implement));
    throw std::runtime_error("[FT Error][MoE Runner] " + err_msg);
  }

  auto init_status = gemm.initialize(args, reinterpret_cast<char *>(workspace) + offset);
  if (init_status != cutlass::Status::kSuccess) {
    std::string err_msg = "Failed to initialize cutlass grouped gemm. Error: " +
                          std::string(cutlassGetStatusString(init_status));
    throw std::runtime_error("[FT Error][MoE Runner] " + err_msg);
  }

  auto run_status = gemm.run(stream);
  if (run_status != cutlass::Status::kSuccess) {
    std::string err_msg = "Failed to run cutlass grouped gemm. Error: " +
                          std::string(cutlassGetStatusString(run_status));
    throw std::runtime_error("[FT Error][MoE Runner] " + err_msg);
  }
}

void nvte_cutlass_grouped_gemm_wgrad(const NVTETensor *A, const NVTETensor *B, NVTETensor *D,
                                     const int64_t *m_splits, const NVTETensor *bias,
                                     NVTETensor *pre_gelu_out, const int num_gemms, bool transa,
                                     bool transb, NVTETensor *workspace, size_t workspaceSize,
                                     bool accumulate, bool use_split_accumulator, int math_sm_count,
                                     hipStream_t stream) {
  NVTE_API_CALL(nvte_cutlass_grouped_gemm_wgrad);
  using namespace transformer_engine;
  //   printf("===========nvte_cutlass_grouped_gemm===========\n");
  //   printf("transa: %d, transb: %d\n", transa, transb);
  //   printf("accumulate: %d\n", accumulate);
  NVTE_CHECK(transa && !transb, "wgrad grouped gemm currently only support TN");

  // Process A
  const transformer_engine::Tensor *inputA = convertNVTETensor(A[0]);
  if (transa) {
    NVTE_CHECK(inputA->has_columnwise_data(), "Input A is missing column-wise usage");
  } else {
    NVTE_CHECK(inputA->has_data(), "Input A is missing row-wise usage");
  }
  __hip_fp8_e4m3_fnuz *inputA_ptr = transa
                                  ? reinterpret_cast<__hip_fp8_e4m3_fnuz *>(inputA->columnwise_data.dptr)
                                  : reinterpret_cast<__hip_fp8_e4m3_fnuz *>(inputA->data.dptr);
  __nv_fp8_e8m0 *inputA_SF_ptr =
      transa ? reinterpret_cast<__nv_fp8_e8m0 *>(inputA->columnwise_scale_inv.dptr)
             : reinterpret_cast<__nv_fp8_e8m0 *>(inputA->scale_inv.dptr);

  // Process B
  const transformer_engine::Tensor *inputB = convertNVTETensor(B[0]);
  if (transb) {
    NVTE_CHECK(inputB->has_data(), "Input B is missing row-wise usage");
  } else {
    NVTE_CHECK(inputB->has_columnwise_data(), "Input B is missing column-wise usage");
  }
  __hip_fp8_e4m3_fnuz *inputB_ptr = transb
                                  ? reinterpret_cast<__hip_fp8_e4m3_fnuz *>(inputB->data.dptr)
                                  : reinterpret_cast<__hip_fp8_e4m3_fnuz *>(inputB->columnwise_data.dptr);
  __nv_fp8_e8m0 *inputB_SF_ptr =
      transb ? reinterpret_cast<__nv_fp8_e8m0 *>(inputB->scale_inv.dptr)
             : reinterpret_cast<__nv_fp8_e8m0 *>(inputB->columnwise_scale_inv.dptr);

  // Process D
  void *outputD_ptr_list[num_gemms];
  for (size_t i = 0; i < num_gemms; i++) {
    const transformer_engine::Tensor *outputD = convertNVTETensor(D[i]);
    NVTE_CHECK(outputD->has_data(), "Input D is missing row-wise usage");
    outputD_ptr_list[i] = outputD->data.dptr;
  }

  // Get GEMM shape
  const int gemm_m = transa ? inputA->flat_last_dim() : inputA->flat_first_dim();
  const int gemm_n = transb ? inputB->flat_first_dim() : inputB->flat_last_dim();
  const int total_gemm_k = transa ? inputA->flat_first_dim() : inputA->flat_last_dim();
  //   printf("num_gemms: %d\n", num_gemms);
  //   printf("gemm_m: %d, gemm_n: %d\n", gemm_m, gemm_n);
  //   printf("total_gemm_k: %d\n", total_gemm_k);
  if ((gemm_m & 0x1F) != 0 || (gemm_n & 0xF) != 0) {
    throw std::runtime_error(
        "gemm_m and gemm_n of grouped gemm with variable K must be multiples of 32.");
  }

  // printf("inputA_SF_ptr: \n");
  // Print_tensor<<<1, 1>>>(inputA_SF_ptr, 128, gemm_k/32);
  // hipDeviceSynchronize();

  //   printf("B_SF_ptr: \n");
  //   Print_tensor<<<1, 1>>>(B_SF_ptr, gemm_n, 256/32);
  //   hipDeviceSynchronize();

  bool transD = true;  // transD should be the same as transB in fprop, currently is always true
  if (transD) {
    if (accumulate) {
      generic_moe_gemm_wgrad_kernelLauncher<__hip_fp8_e4m3_fnuz, __nv_fp8_e8m0, __hip_fp8_e4m3_fnuz,
                                            __nv_fp8_e8m0, float, true>(
          inputA_ptr, inputA_SF_ptr, inputB_ptr, inputB_SF_ptr, outputD_ptr_list, gemm_m, gemm_n,
          m_splits, total_gemm_k, num_gemms, accumulate, workspaceSize,
          convertNVTETensor(workspace[0])->data.dptr, stream);
    } else {
      generic_moe_gemm_wgrad_kernelLauncher<__hip_fp8_e4m3_fnuz, __nv_fp8_e8m0, __hip_fp8_e4m3_fnuz,
                                            __nv_fp8_e8m0, __hip_bfloat16, true>(
          inputA_ptr, inputA_SF_ptr, inputB_ptr, inputB_SF_ptr, outputD_ptr_list, gemm_m, gemm_n,
          m_splits, total_gemm_k, num_gemms, accumulate, workspaceSize,
          convertNVTETensor(workspace[0])->data.dptr, stream);
    }
  } else {
    if (accumulate) {
      generic_moe_gemm_wgrad_kernelLauncher<__hip_fp8_e4m3_fnuz, __nv_fp8_e8m0, __hip_fp8_e4m3_fnuz,
                                            __nv_fp8_e8m0, float, false>(
          inputA_ptr, inputA_SF_ptr, inputB_ptr, inputB_SF_ptr, outputD_ptr_list, gemm_m, gemm_n,
          m_splits, total_gemm_k, num_gemms, accumulate, workspaceSize,
          convertNVTETensor(workspace[0])->data.dptr, stream);
    } else {
      generic_moe_gemm_wgrad_kernelLauncher<__hip_fp8_e4m3_fnuz, __nv_fp8_e8m0, __hip_fp8_e4m3_fnuz,
                                            __nv_fp8_e8m0, __hip_bfloat16, false>(
          inputA_ptr, inputA_SF_ptr, inputB_ptr, inputB_SF_ptr, outputD_ptr_list, gemm_m, gemm_n,
          m_splits, total_gemm_k, num_gemms, accumulate, workspaceSize,
          convertNVTETensor(workspace[0])->data.dptr, stream);
    }
  }
}